
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_SIZE 32 
#define N 10240

__global__ void doubleValues(int*difference, int* numbers, int length) {
  int index = BLOCK_SIZE * blockIdx.x + threadIdx.x;
  if(index != length - 1) {
    difference[index] = numbers[index + 1] - numbers[index]; 
  }
}


int main() {
  int* cpu_arr = (int*)malloc(N * sizeof(int));
  if(!cpu_arr) {
    perror("malloc");
    exit(1);
  }  

  for(int i = 0; i < N; i++) {
    cpu_arr[i] = i * i;
  }

  int* gpu_arr;

  if(hipMalloc(&gpu_arr, sizeof(int) * N) != hipSuccess) {
    fprintf(stderr, "Failed to allocate array on GPU\n");
    exit(2);
  }

  if(hipMemcpy(gpu_arr, cpu_arr, sizeof(int) * N, hipMemcpyHostToDevice) != hipSuccess) {
    fprintf(stderr, "Failed to copy array to the GPU\n");
  }

  int* gpu_difference;
  if(hipMalloc(&gpu_difference, sizeof(int) * N) != hipSuccess) {
    fprintf(stderr, "Failed to allocate array on GPU\n");
    exit(2);
  }

  doubleValues<<<N/BLOCK_SIZE, BLOCK_SIZE>>>(gpu_difference, gpu_arr, N);
  hipDeviceSynchronize();

  if(hipMemcpy(cpu_arr, gpu_difference, sizeof(int) * N, hipMemcpyDeviceToHost) != hipSuccess) {
    fprintf(stderr, "Failed to copy array to the CPU\n");
  }
  
  for(int i = 0; i < N; i++) {
    printf("%d\n", cpu_arr[i]);
  }

  free(cpu_arr);
  hipFree(gpu_arr);
  hipFree(gpu_difference);

  return 0;

}
